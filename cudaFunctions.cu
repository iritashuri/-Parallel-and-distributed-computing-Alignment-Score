#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

#define THREADS_PER_BLOCKS 512

__global__ void AlignmentScore(char *seq1, char *seq2, int seq1_size,
	int seq2_size, float w[4], int max_ms, int max_offset, double *results,
	int num_of_threads);


// Host functions 
void checkErr(hipError_t err, const char *err_message, char *var_name) {
	if (err != hipSuccess) {
		fprintf(stderr, "%s - %s\n, var-> %s", err_message, hipGetErrorString(err), var_name);
		exit(EXIT_FAILURE);
	}
}

void cudaMallocDoubleArr(double** d_arr,size_t arr_size, hipError_t err, char* var_name,const char* malloc_err_message){
	err = hipMalloc((void**)d_arr, arr_size);
	checkErr(err, malloc_err_message, var_name);
}

void cudaMallocFloatArr(float** d_w,size_t w_size, hipError_t err, char* var_name,const char* malloc_err_message){
	err = hipMalloc((void**)d_w, w_size);
	checkErr(err, malloc_err_message, var_name);
}

void cudaMallocString(char** d_seq,size_t seq_size, hipError_t err, char* var_name,const char* malloc_err_message){
	err = hipMalloc((void**)d_seq, seq_size);
	checkErr(err, malloc_err_message, var_name);
}

void cudaMemcpyHostToDeviceFloat(float* d_w, float* w, size_t w_size, hipError_t err,const char* copy_err_message,char* var_name){
	err = hipMemcpy(d_w, w, w_size, hipMemcpyHostToDevice);
	checkErr(err, copy_err_message, var_name);
}

void cudaMemcpyHostToDeviceString(char* d_seq, char* seq, size_t seq_size, hipError_t err,const char* copy_err_message,char* var_name){
	err = hipMemcpy(d_seq, seq, seq_size, hipMemcpyHostToDevice);
	checkErr(err, copy_err_message, var_name);
}

// Call the GPU and get GPU results 
int computeOnGPU(char *seq1, char *seq2, float w[4], int *bestMS, int *best_offset) {
	
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
	char var_name[30];

	// Define results arrays
	double *scores_arr;

	// Define parameters size
	size_t seq1_size = strlen(seq1);
	size_t seq2_size = strlen(seq2);
	int max_offset = (int) seq1_size - (int) seq2_size - 1;
	int max_ms = (int) seq2_size;
	int num_of_threads = (int) max_ms * (max_offset + 1);
	size_t scores_size = num_of_threads * sizeof(double);
	size_t w_size = 4 * sizeof(float);

	// Allocate results arrays in host
	scores_arr = (double*) malloc(scores_size);

	// Allocate memory on GPU to copy the data from the host
	const char *malloc_err_message = "Failed to allocate device memory";
	char *d_seq1;
	char *d_seq2;
	double *d_scores_arr;
	float *d_w;
	
	strcpy(var_name, "d_w");
	cudaMallocFloatArr(&d_w, w_size, err, var_name, malloc_err_message);

	strcpy(var_name, "d_scores_arr");
	cudaMallocDoubleArr(&d_scores_arr, scores_size, err, var_name, malloc_err_message);

	strcpy(var_name, "d_seq1");
	cudaMallocString(&d_seq1, seq1_size * sizeof(char), err, var_name, malloc_err_message);


	strcpy(var_name, "d_seq2");
	cudaMallocString(&d_seq2, seq2_size * sizeof(char), err, var_name, malloc_err_message);	


	// Copy data from host to the GPU memory
	const char *copy_err_message = "Failed to copy data from host to device";
	
	strcpy(var_name, "d_w");
	cudaMemcpyHostToDeviceFloat(d_w, w, w_size, err, copy_err_message, var_name);
	
	strcpy(var_name, "d_seq1");
	cudaMemcpyHostToDeviceString(d_seq1, seq1, seq1_size, err, copy_err_message, var_name);

	strcpy(var_name, "d_seq2");
	cudaMemcpyHostToDeviceString(d_seq2, seq2, seq2_size, err, copy_err_message, var_name);


	// Launch the Kernel
	int num_of_blocks_per_grid = num_of_threads / THREADS_PER_BLOCKS;
	
	// Check if we need to add more block
	if (num_of_threads % THREADS_PER_BLOCKS || !num_of_threads) {
		num_of_blocks_per_grid++;
	}

	AlignmentScore<<<num_of_blocks_per_grid, THREADS_PER_BLOCKS>>>(d_seq1, d_seq2, seq1_size, seq2_size, d_w, max_ms, 		max_offset, d_scores_arr, num_of_threads);
	err = hipGetLastError();

	strcpy(var_name, "No var");
	checkErr(err, "Failed to launch vectorAdd kernel", var_name);

	// Copy the  result from GPU to the host memory.
	const char *copy_res_err_message = "Failed to copy data from decive to host";

	strcpy(var_name, "scores_arr");
	err = hipMemcpy(scores_arr, d_scores_arr, scores_size,
		hipMemcpyDeviceToHost);
	checkErr(err, copy_res_err_message, var_name);

	// Initial bests
	*bestMS = 1;
	*best_offset = 0;
	double best_score = scores_arr[0];
	
        // Compute best score ms and offset
	for (int x = 0; x < num_of_threads; x++) {
		if (scores_arr[x] > best_score) {
			best_score = scores_arr[x];
			*bestMS = x % max_ms + 1;
			*best_offset = x / max_ms;
		}
	}
	
	//printf("bestMS = %d, best_offset = %d, best_score=%lf\n",*bestMS, *best_offset, best_score);	

	free(scores_arr);
	// Free allocated memory on GPU

	const char *free_err_message = "Failed to free device data";
	
	
	strcpy(var_name, "d_scores_arr");
	err = hipFree(d_scores_arr);
	checkErr(err, free_err_message, var_name);

	strcpy(var_name, "d_seq1");
	err = hipFree(d_seq1);
	checkErr(err, free_err_message, var_name);

	strcpy(var_name, "d_seq2");
	err = hipFree(d_seq2);
	checkErr(err, free_err_message, var_name);

	return 0;
}


__device__ void mystrlen(int *len, const char *str) {
	// Calculate length of a string
	(*len) = 0;
	while (*str) {
		(*len)++;
		str++;
	}
}

__device__ void checkspecGroup(const char **group_to_check, int size, char c1, char c2, int *is_cons) {
	// Get group of strings and check if 2 characters are in the same string
	int i, j, k, str_len;
	for (i = 0; i < size; i++) {
		mystrlen(&str_len, group_to_check[i]);
		for (j = 0; j < str_len; j++) {
			if (c1 == group_to_check[i][j]) {
				for (k = 0; k < str_len; k++) {
					if (c2 == group_to_check[i][k]) {
						*(is_cons) = 1;
						return;
					}
				}
			}
		}
	}
}

__device__ void checkConserative(int similarityes[4], char c1, char c2) {
	
	if (c1 == c2) {
		similarityes[0]++;
	} else {
		// If c1 != c2 chack if they in the same Conserative Group and update similarityes[1] if yes
		const char *CONSERVATIVE_GROUPS[9] = { "NDEQ", "MILV", "FYW", "NEQK",
		"QHRK", "HY", "STA", "NHQK", "MILF" };
		const int CONSERVATIVE_GROUPS_SIZE = 9;
		int is_cons = 0;
		checkspecGroup(CONSERVATIVE_GROUPS, CONSERVATIVE_GROUPS_SIZE, c1, c2,
			&is_cons);
		if (is_cons) {
			similarityes[1]++;
		}
		if (!is_cons) {
			// If c1 and c2 are not in the same Conserative Group  check if they in the same Semi Conserative Group
			// And update similarityes[2] if yes
			const char *SEMI_CONSERVATIVE_GROUPS[11] = { "SAG", "SGND",
			"NEQHRK", "ATV", "STPA", "NDEQHK", "HFY", "CSA", "STNK",
			"SNDEQK", "FVLIM" };
			const int SEMI_CONSERVATIVE_GROUPS_SIZE = 11;
			checkspecGroup(SEMI_CONSERVATIVE_GROUPS,
				SEMI_CONSERVATIVE_GROUPS_SIZE, c1, c2, &is_cons);
			if (is_cons) {
				similarityes[2]++;

			}
			// If the not in the same group and not equal update similarityes[3]
			if (!is_cons)
				similarityes[3]++;
		}
	}
}

__device__ void calcSimilarity(char *seq1, char *seq2, int len, int similarityes[4], int ms, int offset) {
	len++;// add 1 to len for the ms
	int i = 0;
	
	// Check if chars in same location (according offset and ms location) are equel, conserative or semi conserative
	// Check chars till ms location
	while (i < ms) {
		checkConserative(similarityes, seq1[i + offset], seq2[i]);
		i++;
	}
	// For ms location is not equel, conserative and semi conserative
	similarityes[3]++;
	i++;
	// Check chars from ms location to the end
	while (i < len) {
		checkConserative(similarityes, seq1[i + offset], seq2[i - 1]);
		i++;
	}
}

__device__ void alignmentScoreFunc(double *results, int similarityes[4], float w[4]) {
	*results = (double) (w[0] * similarityes[0] - w[1] * similarityes[1]
		- w[2] * similarityes[2] - w[3] * similarityes[3]);
}

__global__ void AlignmentScore(char *seq1, char *seq2, int seq1_size,
	int seq2_size, float w[4], int max_ms, int max_offset, double *results,
	int num_of_threads) {
	int new_id = threadIdx.x + (blockDim.x * blockIdx.x);

	if (new_id < num_of_threads) {
		int temp_len;
		// Make sure seq1 and seq2 stay as thet were in file
		mystrlen(&temp_len, seq2);
		if (temp_len > seq2_size)
			seq2[seq2_size] = '\0';
		mystrlen(&temp_len, seq1);
		if (temp_len > seq1_size)
			seq1[seq1_size] = '\0';

		// Cumpure ms and offset to compute
		int my_ms = new_id % max_ms + 1;
		int my_offset = new_id / max_ms;
		// Initial similarityes arr - holds amount of each char in similiarity string
		int similarityes[4] = { 0 };
		
		// Update similarityes arr with amount off each char in similarityes string
		calcSimilarity(seq1, seq2, seq2_size, similarityes, my_ms, my_offset);
		// Compute alignmentScoreFunc
		alignmentScoreFunc(&results[new_id], similarityes, w);
	}
}

